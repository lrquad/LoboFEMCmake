#include "hip/hip_runtime.h"
#include "device_matrix.h"

#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define mylog(token) {std::cout << #token " = " << token << std::endl;}

template <typename T>
hipStream_t device_matrix<T>::_cuda_stream = 0;

template <typename T>
void device_matrix<T>::setCudaStream(hipStream_t& streamId) {
  hipblasSetStream(CUBLAS_HANDLE::getInstance(), streamId);
  _cuda_stream = streamId;
}

// ===============================
// ===== class device_matrix =====
// ===============================
template <typename T>
__global__ void naiveMatrixTranspose(T *odata, const T *idata, const int rows, const int cols) {

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < cols && y < rows)
    odata[x*rows + y] = idata[y*cols+ x];
}
template <typename T> device_matrix<T>::device_matrix():
  _rows(0), _cols(0),
  _capacity(_rows * _cols),
  _data(NULL) { 
}

template <typename T>
device_matrix<T>::device_matrix(size_t r, size_t c):
  _rows(r), _cols(c),
  _capacity(_rows*_cols),
  _data(NULL) {

  _init();
  // Be careful to comment the following line.
  // If the user think the default value are 0, it may give rise to creepy NaN.
  // fillwith(0);
}

template <typename T>
device_matrix<T>::device_matrix(size_t r, size_t c, T value):
  _rows(r), _cols(c),
  _capacity(_rows*_cols),
  _data(NULL) {

  _init();
  fillwith(value);
}

template <typename T>
device_matrix<T>::device_matrix(T* h_data, size_t r, size_t c):
  _rows(r), _cols(c),
  _capacity(_rows*_cols),
  _data(NULL) {

  _init();
  CCE(hipMemcpy(_data, h_data, sizeof(T) * _rows * _cols, hipMemcpyHostToDevice));
}

template <typename T>
device_matrix<T>::device_matrix(const std::string& filename):
  _rows(0), _cols(0),
  _capacity(_rows*_cols),
  _data(NULL) {

  const size_t MAX_BUFFER = 262144;
  char line[MAX_BUFFER];

  FILE* fid = fopen(filename.c_str(), "r");
  while (fgets(line, MAX_BUFFER, fid)) {
    _rows++;

    assert(line[strlen(line) - 1] == '\n');

    if (_cols != 0)
      continue;

    char* token = strtok(line, " \n");
    ++_cols;
    while(strtok(NULL, " \n"))
      ++_cols;
  }
  fseek(fid, 0, SEEK_SET);

  // BEWARE !!
  // BLAS stores data in column-major
  const char *rspecifier = (sizeof(T) / sizeof(float) == 1) ? "%f" : "%lf";

  T* data = new T[_rows*_cols];
  for (size_t i=0; i<_rows; ++i)
    for (size_t j=0; j<_cols; ++j)
      fscanf(fid, rspecifier, &(data[j*_rows + i]));
  fclose(fid);

  _init();

  CCE(hipMemcpy(_data, data, sizeof(T) * _rows * _cols, hipMemcpyHostToDevice));
  delete [] data;
}

// Copy Constructor 
template <typename T>
device_matrix<T>::device_matrix(const device_matrix<T>& source):
  _rows(source._rows), _cols(source._cols),
  _capacity(_rows * _cols),
  _data(NULL) {

  _init();
  CCE(hipMemcpy(_data, source._data, sizeof(T) * _rows * _cols, hipMemcpyDeviceToDevice));
}

template <typename T>
device_matrix<T>::device_matrix(const Transposed& source):
  _rows(source._m._cols), _cols(source._m._rows),
  _capacity(_rows * _cols),
  _data(NULL) {

  _init();
  
  dim3 grid;
  grid.x = (unsigned int) ceil((float) _cols / 32);
  grid.y = (unsigned int) ceil((float) _rows / 32);
  dim3 threads(32, 32);

  naiveMatrixTranspose<<<grid, threads>>>(_data, source._m._data, _rows, _cols);
}

#ifdef HAVE_THRUST_DEVICE_VECTOR_H
// Conversion operator
template <typename T>
device_matrix<T>::operator thrust::device_vector<T>() const {
  assert(_rows == 1 || _cols == 1);
  return thrust::device_vector<T>(_data, _data + size());
}
#endif

template <typename T>
device_matrix<T>::~device_matrix() {
  CudaMemManager<T>::free(_data);
}

// ===========================
// ===== Other Functions =====
// ===========================

// ===== Addition =====
template <typename T>
device_matrix<T>& device_matrix<T>::operator += (T val) {
  cublas_axpy(_rows*_cols, val, SCALAR_MEMORY_BUFFER<T>::getBuffer(), 0, _data, 1);
  return *this;
} 

template <typename T>
device_matrix<T> device_matrix<T>::operator + (T val) const {
  device_matrix<T> m(*this);
  return (m += val);
}

template <typename T>
device_matrix<T>& device_matrix<T>::operator += (const device_matrix<T>& rhs) {
  thrust::device_ptr<T> ptr1(_data);
  thrust::device_ptr<T> ptr2(rhs._data);
  thrust::transform(ptr1, ptr1 + _rows * _cols, ptr2, ptr1, thrust::plus<T>());
  return *this;
}

template <typename T>
device_matrix<T> device_matrix<T>::operator + (const device_matrix<T>& rhs) const {
  device_matrix<T> result(_rows, _cols);

  thrust::device_ptr<T> ptr0(result._data);
  thrust::device_ptr<T> ptr1(_data);
  thrust::device_ptr<T> ptr2(rhs._data);
  thrust::transform(ptr1, ptr1 + _rows * _cols, ptr2, ptr0, thrust::plus<T>());

  return result;
}

template <typename T>
device_matrix<T>& device_matrix<T>::operator += (const typename device_matrix<T>::Transposed& rhs) {
  *this = *this + rhs;
  return *this;
}

template <typename T>
device_matrix<T> device_matrix<T>::operator + (const typename device_matrix<T>::Transposed& rhs) const {
  device_matrix<T> result(_rows, _cols, 0);
  geam(*this, rhs._m, result, (T) 1.0, (T) 1.0, false, true);
  return result;
}

// ===== Substraction =====
template <typename T>
device_matrix<T>& device_matrix<T>::operator -= (T val) {
  val = -val;
  cublas_axpy(_rows*_cols, val, SCALAR_MEMORY_BUFFER<T>::getBuffer(), 0, _data, 1);
  return *this;
}

template <typename T>
device_matrix<T> device_matrix<T>::operator - (T val) const {
  device_matrix<T> m(*this);
  return (m -= val);
}

template <typename T>
device_matrix<T>& device_matrix<T>::operator -= (const device_matrix<T>& rhs) {
  thrust::device_ptr<T> ptr1(_data);
  thrust::device_ptr<T> ptr2(rhs._data);
  thrust::transform(ptr1, ptr1 + _rows * _cols, ptr2, ptr1, thrust::minus<T>());
  return *this;
}

template <typename T>
device_matrix<T> device_matrix<T>::operator - (const device_matrix<T>& rhs) const {
  device_matrix<T> result(_rows, _cols);

  thrust::device_ptr<T> ptr0(result._data);
  thrust::device_ptr<T> ptr1(_data);
  thrust::device_ptr<T> ptr2(rhs._data);
  thrust::transform(ptr1, ptr1 + _rows * _cols, ptr2, ptr0, thrust::minus<T>());

  return result;
}

template <typename T>
device_matrix<T>& device_matrix<T>::operator -= (const typename device_matrix<T>::Transposed& rhs) {
  *this = *this - rhs;
  return *this;
}

template <typename T>
device_matrix<T> device_matrix<T>::operator - (const typename device_matrix<T>::Transposed& rhs) const {
  device_matrix<T> result(_rows, _cols, 0);
  geam(*this, rhs._m, result, (T) 1.0, (T) -1.0, false, true);
  return result;
}

// ===== Division =====
template <typename T>
device_matrix<T>& device_matrix<T>::operator /= (T alpha) {
  return *this *= ( (T) 1 / alpha );
}

template <typename T>
device_matrix<T> device_matrix<T>::operator / (T alpha) const {
  return *this * ( (T) 1 / alpha );
}

// ===== Matrix-scalar Multiplication =====
template <typename T>
device_matrix<T>& device_matrix<T>::operator *= (T alpha) {
  if (alpha != 1)
    cublas_scal(_rows*_cols, alpha, _data, 1);
  return *this;
}

template <typename T>
device_matrix<T> device_matrix<T>::operator * (T alpha) const {
  device_matrix<T> result(*this);
  return result *= alpha;
}

// ===== Matrix-Matrix Multiplication =====
template <typename T>
device_matrix<T>& device_matrix<T>::operator *= (const device_matrix<T>& rhs) {
  *this = *this * rhs;
  return *this;
}

template <typename T>
device_matrix<T> device_matrix<T>::operator * (const device_matrix<T>& rhs) const {
  device_matrix<T> result(_rows, rhs._cols, 0);
  gemm(*this, rhs, result, (T) 1.0, (T) 0.0);
  return result;
}

template <typename T>
device_matrix<T>& device_matrix<T>::operator *= (const Transposed& rhs) {
  *this = *this * rhs;
  return *this;
}

template <typename T>
device_matrix<T> device_matrix<T>::operator * (const Transposed& rhs) const {
  device_matrix<T> result(_rows, rhs._m._rows, 0);
  gemm(*this, rhs._m, result, (T) 1.0, (T) 0.0, false, true);
  return result;
}

// Operator Assignment:
// call copy constructor first, and swap with the temp variable
template <typename T>
device_matrix<T>& device_matrix<T>::operator = (device_matrix<T> rhs) {
  swap(*this, rhs);
  return *this;
}

// Operator transpose
template <typename T>
device_matrix<T>::Transposed device_matrix<T>::operator ~ () const {
  return device_matrix<T>::Transposed(*this);
}

template <typename T>
void device_matrix<T>::_init() {
  _capacity = _rows * _cols;
  _data = CudaMemManager<T>::malloc(_rows * _cols);
}

template <typename T>
void device_matrix<T>::resize(size_t r, size_t c) {
  // printf("trying to resize from (%lu, %lu) => (%lu, %lu), with original capacity = %lu\n", _rows, _cols, r, c, _capacity);
  if (_rows == r && _cols == c)
    return;

  _rows = r;
  _cols = c;

  if (r * c <= _capacity)
    return;

  CudaMemManager<T>::free(_data);
  _init();
}

template <typename T>
void device_matrix<T>::resize(size_t r, size_t c, T value) {
  this->resize(r, c);
  fillwith(value);
}

template <typename T>
void device_matrix<T>::reserve(size_t capacity) {
  if (capacity <= _capacity)
    return;

  _capacity = capacity;

  T* buffer = CudaMemManager<T>::malloc(_capacity);
  CCE(hipMemcpy(buffer, _data, sizeof(T) * size(), hipMemcpyDeviceToDevice));
  CudaMemManager<T>::free(_data);
  _data = buffer;
}

template <typename T>
void device_matrix<T>::print(FILE* fid, int precision, char delimiter) const {

  if (_rows == 0 || _cols == 0)
    return;

  T* data = new T[size()];
  CCE(hipMemcpy(data, _data, sizeof(T) * size(), hipMemcpyDeviceToHost));

  char format[16];
  sprintf(format, "%c%%.%de", delimiter, precision < 0 ? 0 : precision);

  for (size_t i=0; i<_rows; ++i) {
    fprintf(fid, format, data[i]);
    for (size_t j=1; j<_cols; ++j)
      fprintf(fid, format, data[j*_rows + i]);
    fprintf(fid, "\n");
  }

  delete [] data;
}

template <typename T>
void device_matrix<T>::fillwith(T val) {
  hipMemset(_data, 0, _rows * _cols * sizeof(T));

  if (val != 0)
    *this += val;
}

template <typename T>
void device_matrix<T>::save(const std::string& filename) const {
  FILE* fid = fopen(filename.c_str(), "w");
  if (fid == NULL)
    return;

  print(fid);
  fclose(fid);
}

template <>
void device_matrix<float>::cublas_gemm(
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k,
  float alpha,
  const float* A, int lda,
  const float* B, int ldb,
  float beta,
  float* C, int ldc) {
  CCE(hipblasSgemm(CUBLAS_HANDLE::getInstance(), transA, transB, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc));
}

template <>
void device_matrix<double>::cublas_gemm(
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k,
  double alpha,
  const double* A, int lda,
  const double* B, int ldb,
  double beta,
  double* C, int ldc) {
  CCE(hipblasDgemm(CUBLAS_HANDLE::getInstance(), transA, transB, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc));
}

template <>
void device_matrix<float>::cublas_geam(
    hipblasOperation_t transA, hipblasOperation_t transB,
    int m, int n,
    float alpha, const float *A, int lda,
    float beta , const float *B, int ldb,
    float *C, int ldc) {
  CCE(hipblasSgeam(CUBLAS_HANDLE::getInstance(), transA, transB, m, n, &alpha, A, lda, &beta, B, ldb, C, ldc));
}

template <>
void device_matrix<double>::cublas_geam(
    hipblasOperation_t transA, hipblasOperation_t transB,
    int m, int n,
    double alpha, const double *A, int lda,
    double beta , const double *B, int ldb,
    double *C, int ldc) {
  CCE(hipblasDgeam(CUBLAS_HANDLE::getInstance(), transA, transB, m, n, &alpha, A, lda, &beta, B, ldb, C, ldc));
}

template <>
void device_matrix<float>::cublas_gemv(
    hipblasOperation_t trans,
    int m, int n,
    float alpha,
    const float *A, int lda,
    const float *x, int incx,
    float beta,
    float *y, int incy) {
  CCE(hipblasSgemv(CUBLAS_HANDLE::getInstance(), trans, m, n, &alpha, A, lda, x, incx, &beta, y, incy));
}

template <>
void device_matrix<double>::cublas_gemv(
    hipblasOperation_t trans,
    int m, int n,
    double alpha,
    const double *A, int lda,
    const double *x, int incx,
    double beta,
    double *y, int incy) {
  CCE(hipblasDgemv(CUBLAS_HANDLE::getInstance(), trans, m, n, &alpha, A, lda, x, incx, &beta, y, incy));
}

template <>
void device_matrix<float>::cublas_dot(int n,const float *X, int incx,
  const float *Y, int incy,float* C)
  {
    CCE(hipblasSdot(CUBLAS_HANDLE::getInstance(),n,X,incx,Y,incy,C));
  }

template <>
void device_matrix<double>::cublas_dot(int n,const double *X, int incx,
  const double *Y, int incy,double* C)
  {
    CCE(hipblasDdot(CUBLAS_HANDLE::getInstance(),n,X,incx,Y,incy,C));
  }

template <>
void device_matrix<float>::cublas_iamax(int n, const float *x, int incx, int *result) {
  CCE(hipblasIsamax(CUBLAS_HANDLE::getInstance(), n, x, incx, result));
}

template <>
void device_matrix<double>::cublas_iamax(int n, const double *x, int incx, int *result) {
  CCE(hipblasIdamax(CUBLAS_HANDLE::getInstance(), n, x, incx, result));
}


template <>
void device_matrix<float>::cublas_nrm2(int n, const float *x, int incx, float *result) {
  CCE(hipblasSnrm2(CUBLAS_HANDLE::getInstance(), n, x, incx, result));
}

template <>
void device_matrix<double>::cublas_nrm2(int n, const double *x, int incx, double *result) {
  CCE(hipblasDnrm2(CUBLAS_HANDLE::getInstance(), n, x, incx, result));
}

template <>
void device_matrix<float>::cublas_scal(int n, float alpha, float *x, int incx) {
  CCE(hipblasSscal(CUBLAS_HANDLE::getInstance(), n, &alpha, x, incx));
}

template <>
void device_matrix<double>::cublas_scal(int n, double alpha, double *x, int incx) {
  CCE(hipblasDscal(CUBLAS_HANDLE::getInstance(), n, &alpha, x, incx));
}

template <>
void device_matrix<float>::cublas_axpy(
    int n, float alpha,
    const float *x, int incx,
    float *y, int incy) {
  CCE(hipblasSaxpy(CUBLAS_HANDLE::getInstance(), n, &alpha, x, incx, y, incy));
}

template <>
void device_matrix<double>::cublas_axpy(
    int n, double alpha,
    const double *x, int incx,
    double *y, int incy) {
  CCE(hipblasDaxpy(CUBLAS_HANDLE::getInstance(), n, &alpha, x, incx, y, incy));
}

// ++++++++++++++++++++++++++++++++++++++++++++
// +++++ Template Explicit Initialization +++++
// ++++++++++++++++++++++++++++++++++++++++++++
template class device_matrix<float>;
template class device_matrix<double>;
